#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <glm\glm.hpp>
#include <gl\GL.h>
#include <qt\qdebug.h>

#include "DeepLearner.h"
#include <random>
#include <time.h>
#include <algorithm>
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "StateAction.h"
#include "CudaCode\ScreenManipulation.cu"

using std::vector;

#pragma region Initialization
void DeepLearner::Initialize(int* scorePoint, int* widthPoint, int* heightPoint, int Number_Of_Inputs, float learningRate, type algoType){
	score = scorePoint;
	width = widthPoint;
	rWidth = *width / 2;
	//qDebug() << "Width: " << width << endl << "Reduce Width: " << rWidth << endl;
	height = heightPoint;
	rHeight = *height / 2;
	//qDebug() << "height: " << height << endl << "Reduce height: " << rHeight << endl;
	numInput = Number_Of_Inputs;
	algo = algoType;
	lr = learningRate;
	srand(time(NULL));
	numCalls = 0;
	lastInput = 0;
	reduceScreen = new float[rWidth * rHeight];
	inputWeights = new float[50 * 50];
	for (int i = 0; i < 50 * 50; ++i){
		inputWeights[i] = rand.randomFloat();
	}

	//Number of hidden nodes * number of input nodes
	firstHiddenWeights = new float[10 * (50*50)];
	for (int i = 0; i < 10 * 50 * 50; ++i){
		firstHiddenWeights[i] = rand.randomFloat();
	}
	
	bias = new float[10];
	for (int i = 0; i < 10; ++i){
		bias[i] = -1 * rand.randomInRange(0, 40.0f);
	}

	outputWeights = new float[10 * numInput];
	for (int i = 0; i < 10 * numInput; ++i){
		outputWeights[i] = rand.randomFloat();
	}
}

DeepLearner::DeepLearner() : f_RandomChance(0.1) 
{

}

DeepLearner::~DeepLearner()
{
	//free(reduceScreen);
}
#pragma endregion

#pragma region Cuda Code
////This is where I get calculations, and pass the screengrab down to the neurons.
//Original don't wanna delete yet.
//__global__ void CalcInput(float* screen, int* d_Input, int* d_numInput){
//	//printf("Test\n");
//	int id = threadIdx.x + blockDim.x * blockIdx.x;
//
//	int intensity;
//	intensity = (screen[80314] * 100.0f);
//	*d_Input = intensity % *d_numInput;// *d_numInput - 1;
//}

//__global__ void updateInput(float* screen, float* weight, float* )
__global__ void CalcInput(float* screen, float* weight, float* d_Votes){
	int id = threadIdx.x + blockDim.x * blockIdx.x;

	d_Votes[id] = screen[id] * weight[id];
}

__global__ void FirstHidden(float* input, float* weight, float* bias, int d_numVotes, int* d_votes){
	int id = threadIdx.x + blockDim.x * blockIdx.x;

	float total = 0.0f;

	//printf("Num Votes: %i", d_numVotes);

	for (int i = 0; i < d_numVotes; ++i){
		//if (weight[id*d_numVotes + i] > 0) printf("Weight higher than 0: %f", weight[id*d_numVotes + i]);
		//if (input[i] > 0) printf("Input: %f ", input[i]);
		//printf("Weight: %f\n", weight[id * d_numVotes + i]);

		total += input[i] * weight[id * d_numVotes + i];
	}

	//Should use sigmoid here. Maybe Could be in for loop though
	total += *bias;
	//printf("Total: %f\n", total);
	total = (int)(1 / (1 + exp(-total))) % 3;
	//total = ((int)(total)) % 3;

	//printf("Total: %f\n", total);
	d_votes[id] = total;
}

__global__ void OutputLayer(float* hiddenVotes, float* weight, int d_numHiddenNodes, float* d_votes){
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	
	d_votes[id] = 0.0f;

	for (int i = 0; i < d_numHiddenNodes; ++i){
		//printf("Hidden Votes: %f, Weight: %f\n", hiddenVotes[i], weight[id * d_numHiddenNodes + i]);
		d_votes[id] += hiddenVotes[i] * weight[id * d_numHiddenNodes + i];
	}

	printf("Votes: %f\n", d_votes[id]);
}

__global__ void GreyScreen(float* d_pixelsR, float* d_pixelsG, float* d_pixelsB,
	float* d_reducePixels, int numPixels){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("Test ID: %u ", numPixels);
	if (id < numPixels){
		d_reducePixels[id] = (d_pixelsR[id] + d_pixelsG[id] + d_pixelsB[id]) / 3;
		//printf("Reduce Pixels ");
		//printf("%f ", d_reducePixels[id]);
	}
}
#pragma endregion

#pragma region input
//Find what input would be best.
int  DeepLearner::GetInput(vector<float*> screengrab){
	numCalls++;
	if (numCalls > 3){
#pragma region Random Input
		if (rand.randomInRange(0, 1) < f_RandomChance){
			lastInput = rand.randomInRange(0, numInput);
			numCalls = 0;
		}
#pragma endregion

		else{

//Screen maniputlation only works for 800x600 screens currently. Changes to a screen of 400x300 greyscaled
//Also stores the 8x6 mini pixel set in the variable screenbits
#pragma region screen manipulation
			GetScreen();
			//Seperate the reduce screen into 8x6 chunks.
			//Average the intensity for those pixels.
			float* screenBits = new float[50 * 50];
			int bitsIndex = 0;

			//Reduce the screen into a 50x50 grid of 8x6 pixels. Average the intensity to get the average brightness of that grid.
			for (int r = 0; r < 50; r++){
				for (int c = 0; c < 50; c++){
					float intense = 0.0f;
					int numPixels = 0;
					for (int row = 0; row < 6; ++row){
						for (int col = 0; col < 8; ++col){
							float reduxIntense = reduceScreen[(r * 400 * 6) + (row * 400) + (c * 8 + col)];
							intense += reduxIntense;
							++numPixels;
						}
					}
					if (intense > 0.0f){
						//qDebug() << "Intense value: " << intense;
					}
					screenBits[bitsIndex] = (intense) / ((float)numPixels);
					++bitsIndex;
				}
			}

#pragma endregion

//Get all the inputs times their weight and store it in the array InputVotes
#pragma region Input Weights
			float* d_screen;
			float* d_weights;
			int* d_numInput;
			float* d_Votes;
			float* InputVotes = new float[4 * 625];

			int sizeInt = sizeof(int);
			int sizeScreen = (50 * 50) *sizeof(float);
			int sizeWeights = (50 * 50) *sizeof(int);

			hipMalloc((void**)&d_screen, sizeScreen);
			hipMalloc((void**)&d_weights, sizeWeights);
			hipMalloc((void**)&d_Votes, sizeWeights);
			hipMalloc((void**)&d_numInput, sizeInt);

			hipMemcpy(d_screen, screenBits, sizeScreen, hipMemcpyHostToDevice);
			hipMemcpy(d_weights, inputWeights, sizeWeights, hipMemcpyHostToDevice);
			hipMemcpy(d_numInput, &numInput, sizeInt, hipMemcpyHostToDevice);

			CalcInput <<< 4, 625 >>>(d_screen, d_weights, d_Votes);

			hipMemcpy(InputVotes, d_Votes, sizeWeights, hipMemcpyDeviceToHost);

			hipFree(d_Votes);
			hipFree(d_numInput);
			hipFree(d_screen);
			hipFree(d_numInput);

#pragma endregion

//Run sigmoid on all input and store the votes or output in the array HiddenVotes
#pragma region FirstHidden
			float* d_InputVotes;
			float* d_FHW;
			float* d_bias;
			int* d_HiddenVotes;
			int* HiddenVotes;

			int sizeFHW = (10 * 50 * 50) *sizeof(float);
			int sizeHidden = 10 * sizeof(float);
			int sizeInputVotes = 50 * 50 * sizeof(float);
			int sizeBias = 10 * sizeof(float);

			hipMalloc((void**)&d_FHW, sizeFHW);
			hipMalloc((void**)&d_HiddenVotes, sizeHidden);
			hipMalloc((void**)&d_InputVotes, sizeInputVotes);
			hipMalloc((void**)&d_bias, sizeBias);
			HiddenVotes = new int[10];

			hipMemcpy(d_FHW, firstHiddenWeights, sizeFHW, hipMemcpyHostToDevice);
			hipMemcpy(d_InputVotes, InputVotes, sizeInputVotes, hipMemcpyHostToDevice);
			hipMemcpy(d_bias, bias, sizeBias, hipMemcpyHostToDevice);

			//Input votes, Hidden weights, Number of Inputs, Votes array
			FirstHidden <<<1, 10 >>>(d_InputVotes, d_FHW, d_bias, 50 * 50, d_HiddenVotes);

			hipMemcpy(HiddenVotes, d_HiddenVotes, sizeHidden, hipMemcpyDeviceToHost);
			
			hipFree(d_InputVotes); hipFree(d_FHW); hipFree(d_HiddenVotes);

#pragma endregion

//Connect all hidden nodes to the output nodes. Store values in the array votes
#pragma region output
			float* d_outputHiddenVotes;
			float* d_outputWeights;
			float* d_votes;
			float* votes;

			votes = new float[3];
			
			int sizeHiddenOutput = 10 * sizeof(float);
			int sizeWeightsOutput = 10 * numInput * sizeof(float);
			int sizeVotesOutput = numInput * sizeof(float);

			hipMalloc((void**)&d_outputHiddenVotes, sizeHiddenOutput);
			hipMalloc((void**)&d_outputWeights, sizeWeightsOutput);
			hipMalloc((void**)&d_votes, sizeVotesOutput);

			hipMemcpy(d_outputHiddenVotes, HiddenVotes, sizeHiddenOutput, hipMemcpyHostToDevice);
			hipMemcpy(d_outputWeights, outputWeights, sizeWeightsOutput, hipMemcpyHostToDevice);

			//The number of threads is the number of inputs possible, so Left or Right
			//The third varaible is the number of hidden layers
			OutputLayer <<<1, 3 >>>(d_outputHiddenVotes, d_outputWeights, 10, d_votes);

			hipMemcpy(votes, d_votes, sizeVotesOutput, hipMemcpyDeviceToHost);

			hipFree(d_outputHiddenVotes); hipFree(d_outputWeights); hipFree(d_votes);
#pragma endregion

#pragma region Tally Votes
			//TODO: tally is currently a memory leak. I should fix when I can
			int tally = 0.0f;
			for (int i = 0; i < numInput; ++i){
				qDebug() << "Input: " << i << " Tally: " << votes[i];
				if (tally < votes[i]){
					tally = votes[i];
					lastInput = i;
				}
			}

			qDebug();
			qDebug();

			delete[] screenBits;
			//delete[] tally;
			delete[] InputVotes;
			delete[] HiddenVotes;
			delete[] votes;
			numCalls = 0;
#pragma endregion

#pragma region First CalcInput code
			//It works, don't wanna delete
			//float* d_screen = screenBits;
			//int* d_Input = &lastInput;
			////std::cout << *d_Input << std::endl;
			//int* d_numInput = &numInput;
			////std::cout << *d_numInput << std::endl;
			//int sizeInt = sizeof(int);
			//int sizeArray = (rWidth * rHeight) * sizeof(float) / 4;
			//qDebug() << "Array size: " << sizeArray / 4 << endl;
			//hipMalloc((void**)&d_Input, sizeInt);
			//hipMalloc((void**)&d_numInput, sizeInt);
			//hipMalloc((void**)&d_screen, sizeArray);
			//hipMemcpy(d_numInput, &numInput, sizeInt, hipMemcpyHostToDevice);
			//hipMemcpy(d_screen, reduceScreen, sizeArray, hipMemcpyHostToDevice);
			//CalcInput << <1, 1 >> >(d_screen, d_Input, d_numInput);
			//hipMemcpy(&lastInput, d_Input, sizeInt, hipMemcpyDeviceToHost);
			////std::cout << *d_Input << std::endl;
			////std::cout << *d_numInput << std::endl;
			//hipFree(d_Input);
			//hipFree(d_numInput);
			//hipFree(d_screen);
			//numCalls = 0;
			////free(d_Input); free(d_numInput);
#pragma endregion
		}
	}
	//Store state action pairs
	//Seenms like an array of 200-250 values is what I have with full screen.
	//Multiple arrays didn't work either. I'm seriously just limited in how many I get...

	return lastInput;
}

void DeepLearner::GetScreen(){
	glReadBuffer(GL_FRONT);
	int numPixels = *width * *height;
	GLfloat* pixelsR = new GLfloat[numPixels];
	GLfloat* pixelsG = new GLfloat[numPixels];
	GLfloat* pixelsB = new GLfloat[numPixels];
	glReadPixels(0, 0, *width, *height, GL_RED, GL_FLOAT, pixelsR);
	glReadPixels(0, 0, *width, *height, GL_GREEN, GL_FLOAT, pixelsG);
	glReadPixels(0, 0, *width, *height, GL_BLUE, GL_FLOAT, pixelsB);
	float* greyScreen = new float[numPixels];

#pragma region Serial Implementation

	//Greyscale the image
	for (int i = 0; i<numPixels; ++i){
		greyScreen[i] = 0.144*pixelsR[i] + 0.587*pixelsG[i] + 0.299*pixelsB[i];
	}

	//for (int i = 0; i < numPixels; i++){
	//	if (greyScreen[i] > 0.0f){
	//		qDebug() << "Pixel: " << i << " Intensity: " << greyScreen[i];
	//	}
	//}

	//Shrink the image
	int i = 0;
	for (int c = 0; c < *height; c += 2){
		for (int r = 0; r < *width; r += 2){
			float x1 = greyScreen[r + c * 800];
			float x2 = greyScreen[r + 1 + c * 800];
			float x3 = greyScreen[r + (c + 1) * 800];
			float x4 = greyScreen[r + 1 + (c + 1)  * 800];

			float avg = (x1 + x2 + x3 + x4) / 4;

			reduceScreen[i] = avg;
			++i;
		}
	}

	//for (int i = 0; i < *width/2 * *height/2; i++){
	//	if (reduceScreen[i] > 0.0f){
	//		qDebug() << "Pixel: " << i << " Intensity: " << reduceScreen[i];
	//	}
	//}
#pragma endregion

#pragma region CUDA implementation, currently broke
	//float* d_pixelsR;
	//float* d_pixelsG;
	//float* d_pixelsB;
	//float* d_reducePixels;
	//hipMalloc((void**)&d_pixelsR, numPixels);
	//hipMalloc((void**)&d_pixelsG, numPixels);
	//hipMalloc((void**)&d_pixelsB, numPixels);
	//hipMalloc((void**)&d_reducePixels, numPixels);

	//hipMemcpy(d_pixelsR, pixelsR, numPixels, hipMemcpyHostToDevice);
	//hipMemcpy(d_pixelsG, pixelsG, numPixels, hipMemcpyHostToDevice);
	//hipMemcpy(d_pixelsB, pixelsB, numPixels, hipMemcpyHostToDevice);
	////qDebug() << "It is running";
	////qDebug() << "Num Pixels / 1024" << numPixels / 1024 << endl;
	//GreyScreen <<<1, 1024>>> (d_pixelsR, d_pixelsG, d_pixelsB, d_reducePixels, numPixels);

	////hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	////GreyScreen <<<1, numPixels >>>(d_pixelsR, d_pixelsG, d_pixelsB, d_reducePixels, numPixels);
	//hipMemcpy(greyScreen, d_reducePixels, numPixels, hipMemcpyDeviceToHost);

	//for (int i = 0; i < numPixels; i++){
	//	if (greyScreen[i] > 0.0f){
	//		//qDebug() << "Pixel: " << i << " Intensity: " << greyScreen[i];
	//	}
	//}

	//hipFree(d_reducePixels); 
	//hipFree(d_pixelsR); 
	//hipFree(d_pixelsG); 
	//hipFree(d_pixelsB);
	
#pragma endregion

	free(pixelsR); free(pixelsG); free(pixelsB); free(greyScreen);
	//qDebug() << "Got a screen, mebe";
}

void DeepLearner::GameOver(bool isWin){
	//Modify weights to decrease the value of what happened.
}

void DeepLearner::SwitchAlgorithm(type algoType){
	//Changes the algorith, may remove.
	algo = algoType;
}

void DeepLearner::learn(){
	//Keep practicing Games over and over
}

void DeepLearner::play(){
	//Play a game with a lr of 0;
}

#pragma endregion