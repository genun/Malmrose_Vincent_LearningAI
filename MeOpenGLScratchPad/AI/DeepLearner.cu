#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <glm\glm.hpp>
#include <gl\GL.h>
#include <qt\qdebug.h>

#include "DeepLearner.h"
#include <random>
#include <time.h>
#include <algorithm>
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "StateAction.h"
#include "CudaCode\ScreenManipulation.cu"

using std::vector;

#pragma region Initialization
void DeepLearner::Initialize(int* scorePoint, int* widthPoint, int* heightPoint, int Number_Of_Inputs, float learningRate, type algoType){
	score = scorePoint;
	width = widthPoint;
	rWidth = *width / 2;
	qDebug() << "Width: " << width << endl << "Reduce Width: " << rWidth << endl;
	height = heightPoint;
	rHeight = *height / 2;
	qDebug() << "height: " << height << endl << "Reduce height: " << rHeight << endl;
	numInput = Number_Of_Inputs;
	algo = algoType;
	lr = learningRate;
	srand(time(NULL));
	numCalls = 0;
	lastInput = 0;
	reduceScreen = new float[rWidth * rHeight];
}

DeepLearner::DeepLearner()
{

}

DeepLearner::~DeepLearner()
{
	//free(reduceScreen);
}
#pragma endregion

#pragma region Cuda Code
//This is where I get calculations, and pass the screengrab down to the neurons.
__global__ void CalcInput(float* screen, int* d_Input, int* d_numInput){
	//printf("Test\n");
	int id = threadIdx.x + blockDim.x * blockIdx.x;

	int intensity;
	intensity = (screen[80314] * 100.0f);
	*d_Input = intensity % *d_numInput;// *d_numInput - 1;
}

__global__ void GreyScreen(float* d_pixelsR, float* d_pixelsG, float* d_pixelsB,
	float* d_reducePixels, int numPixels){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("Test ID: %u ", numPixels);
	if (id < numPixels){
		d_reducePixels[id] = (d_pixelsR[id] + d_pixelsG[id] + d_pixelsB[id]) / 3;
		//printf("Reduce Pixels ");
		//printf("%f ", d_reducePixels[id]);
	}
}
#pragma endregion

#pragma region input
//Find what input would be best.
int  DeepLearner::GetInput(vector<float*> screengrab){
	numCalls++;
	if (numCalls > 3){
		if (rand.randomInRange(0, 1) < f_RandomChance){
			lastInput = rand.randomInRange(0, numInput);
		}
		else{
			GetScreen();

			float* d_screen = reduceScreen;
			int* d_Input = &lastInput;
			//std::cout << *d_Input << std::endl;
			int* d_numInput = &numInput;
			//std::cout << *d_numInput << std::endl;
			int sizeInt = sizeof(int);
			int sizeArray = (rWidth * rHeight) * sizeof(float) / 4;
			qDebug() << "Array size: " << sizeArray /4 << endl;
			hipMalloc((void**)&d_Input, sizeInt);
			hipMalloc((void**)&d_numInput, sizeInt);
			hipMalloc((void**)&d_screen, sizeArray);
			hipMemcpy(d_numInput, &numInput, sizeInt, hipMemcpyHostToDevice);
			hipMemcpy(d_screen, reduceScreen, sizeArray, hipMemcpyHostToDevice);
			CalcInput <<<1, 1>>>(d_screen, d_Input, d_numInput);
			hipMemcpy(&lastInput, d_Input, sizeInt, hipMemcpyDeviceToHost);
			//std::cout << *d_Input << std::endl;
			//std::cout << *d_numInput << std::endl;
			hipFree(d_Input);
			hipFree(d_numInput);
			hipFree(d_screen);
			numCalls = 0;
		//free(d_Input); free(d_numInput);
		}
	}
	//Store state action pairs
	//Seenms like an array of 200-250 values is what I have with full screen.
	//Multiple arrays didn't work either. I'm seriously just limited in how many I get...

	return lastInput;
}

void DeepLearner::GetScreen(){
	glReadBuffer(GL_FRONT);
	int numPixels = *width * *height;
	GLfloat* pixelsR = new GLfloat[numPixels];
	GLfloat* pixelsG = new GLfloat[numPixels];
	GLfloat* pixelsB = new GLfloat[numPixels];
	glReadPixels(0, 0, *width, *height, GL_RED, GL_FLOAT, pixelsR);
	glReadPixels(0, 0, *width, *height, GL_GREEN, GL_FLOAT, pixelsG);
	glReadPixels(0, 0, *width, *height, GL_BLUE, GL_FLOAT, pixelsB);
	float* greyScreen = new float[numPixels];
#pragma region Serial Implementation

	//Greyscale the image
	for (int i = 0; i<numPixels; ++i){
		greyScreen[i] = 0.144*pixelsR[i] + 0.587*pixelsG[i] + 0.299*pixelsB[i];
	}

	//for (int i = 0; i < numPixels; i++){
	//	if (greyScreen[i] > 0.0f){
	//		qDebug() << "Pixel: " << i << " Intensity: " << greyScreen[i];
	//	}
	//}

	//Shrink the image
	int i = 0;
	for (int c = 0; c < *height; c += 2){
		for (int r = 0; r < *width; r += 2){
			float x1 = greyScreen[r + c * 800];
			float x2 = greyScreen[r + 1 + c * 800];
			float x3 = greyScreen[r + (c + 1) * 800];
			float x4 = greyScreen[r + 1 + (c + 1)  * 800];

			float avg = (x1 + x2 + x3 + x4) / 4;

			reduceScreen[i] = avg;
			++i;
		}
	}

	//for (int i = 0; i < *width/2 * *height/2; i++){
	//	if (reduceScreen[i] > 0.0f){
	//		qDebug() << "Pixel: " << i << " Intensity: " << reduceScreen[i];
	//	}
	//}
#pragma endregion

#pragma region CUDA implementation, currently broke
	//float* d_pixelsR;
	//float* d_pixelsG;
	//float* d_pixelsB;
	//float* d_reducePixels;
	//hipMalloc((void**)&d_pixelsR, numPixels);
	//hipMalloc((void**)&d_pixelsG, numPixels);
	//hipMalloc((void**)&d_pixelsB, numPixels);
	//hipMalloc((void**)&d_reducePixels, numPixels);

	//hipMemcpy(d_pixelsR, pixelsR, numPixels, hipMemcpyHostToDevice);
	//hipMemcpy(d_pixelsG, pixelsG, numPixels, hipMemcpyHostToDevice);
	//hipMemcpy(d_pixelsB, pixelsB, numPixels, hipMemcpyHostToDevice);
	////qDebug() << "It is running";
	////qDebug() << "Num Pixels / 1024" << numPixels / 1024 << endl;
	//GreyScreen <<<1, 1024>>> (d_pixelsR, d_pixelsG, d_pixelsB, d_reducePixels, numPixels);

	////hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	////GreyScreen <<<1, numPixels >>>(d_pixelsR, d_pixelsG, d_pixelsB, d_reducePixels, numPixels);
	//hipMemcpy(greyScreen, d_reducePixels, numPixels, hipMemcpyDeviceToHost);

	//for (int i = 0; i < numPixels; i++){
	//	if (greyScreen[i] > 0.0f){
	//		//qDebug() << "Pixel: " << i << " Intensity: " << greyScreen[i];
	//	}
	//}

	//hipFree(d_reducePixels); 
	//hipFree(d_pixelsR); 
	//hipFree(d_pixelsG); 
	//hipFree(d_pixelsB);
	
#pragma endregion

	free(pixelsR); free(pixelsG); free(pixelsB); free(greyScreen);
	//qDebug() << "Got a screen, mebe";
}

void DeepLearner::GameOver(bool isWin){
	//Modify weights to decrease the value of what happened.
}

void DeepLearner::SwitchAlgorithm(type algoType){
	//Changes the algorith, may remove.
	algo = algoType;
}

void DeepLearner::learn(){
	//Keep practicing Games over and over
}

void DeepLearner::play(){
	//Play a game with a lr of 0;
}

#pragma endregion